
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void whoami(void)
{
    
    int block_id = blockIdx.x +
                   blockIdx.y * gridDim.x +
                   blockIdx.z * gridDim.x * gridDim.y;

    int block_offset = block_id *
                       blockDim.x * blockDim.y * blockDim.z;

    int thread_offset = threadIdx.x +
                        threadIdx.y * blockDim.x +
                        threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset;

    printf("%04d | Block (%d %d %d) = %3d | Thread (%d %d %d) = %3d\n",
           id,
           blockIdx.x, blockIdx.y, blockIdx.z, block_id,
           threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
}

int main(int argc, char **argv)
{
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4;

    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    std::cout << blocks_per_grid << " blocks/grid" << std::endl;
    std::cout << threads_per_block << " threads/block" << std::endl;
    std::cout << threads_per_block * blocks_per_grid << " total threads" << std::endl;

    dim3 blocksPerGrid(b_x, b_y, b_z);
    dim3 threadsPerBlock(t_x, t_y, t_z);

    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
}